#include "hip/hip_runtime.h"
#include <stdio.h>
#include "image.h"
#include <time.h>
#define SIZE 2048*1024
#define BLOCKS 12

__global__ void histo_MultiBlock( unsigned char *buffer,long size,unsigned int *histo ) {

__shared__ unsigned int temp[256];
int i = threadIdx.x + blockIdx.x * blockDim.x;
int offset= blockDim.x * gridDim.x;
int memoffset = blockIdx.x * blockDim.x;
temp[threadIdx.x] = 0;
__syncthreads();

while(i<size){
	
	atomicAdd( &temp[buffer[i]], 1);
	i+=offset;
}
__syncthreads();
atomicAdd( &(histo[threadIdx.x+memoffset]), temp[threadIdx.x] );
}

int main(void){

unsigned char *dev_buffer;
unsigned int *dev_histo;

//pedimos memoria
hipMalloc( (void**)&dev_buffer, SIZE );
hipMemcpy( dev_buffer, image, SIZE, hipMemcpyHostToDevice );

hipMalloc( (void**)&dev_histo,256 * BLOCKS * sizeof(long) );
hipMemset( dev_histo, 0, 256*BLOCKS *sizeof(int) );
//KERNEL EXECUTION
histo_MultiBlock<<<BLOCKS,256>>>(dev_buffer,SIZE,dev_histo);

//KERNEL HAS FINISHED 
unsigned int host_histo[BLOCKS][256];
//retorno de valores desde dev a host
hipMemcpy( host_histo, dev_histo, 256*BLOCKS*sizeof(int),hipMemcpyDeviceToHost );
/*
for(long i=0;i<256;i++){
	for(int j=1;j<BLOCKS;j++){
		printf("%d ",host_histo[j][i]);			
	}
}
printf("\n\n");*/
//unimos histogramas
for(long i=0;i<256;i++){
	for(int j=1;j<BLOCKS;j++){
		host_histo[0][i]+=host_histo[j][i];			
	}
}
//calculamos si el histograma es correcto en CPU, debemos obtener 0 (restamos en vez de sumar uno)
for(int i=0;i<256;i++){
	printf("%d ",host_histo[0][i]);
}
for(int i=0;i<SIZE;i++){
	host_histo[0][image[i]]--;
}
printf("\n\n");
//buscamos valores distintos de 0
for(int i=0;i<256;i++){
	if(host_histo[0][i]!=0){
		printf("valor %d incorrecto\n",i);
		break;
	}
}
//liberamos memoria
hipFree(dev_histo);
hipFree(dev_buffer);

return 0;
}
