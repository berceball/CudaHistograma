#include "hip/hip_runtime.h"
#include <stdio.h>
#include "image.h"
#include <time.h>
#define SIZE 2048*1024
#define BLOCKS 1000
#define THREADS 256
__global__ void histo_MultiBlock( unsigned char *buffer,long size,unsigned int *histo ) {

	__shared__ unsigned int temp[256];
	int i = threadIdx.x + blockIdx.x * THREADS;
	int offset= THREADS * BLOCKS;
	int memoffset = blockIdx.x * THREADS;
if(threadIdx.x <256)
	temp[threadIdx.x] = 0;
	__syncthreads();

	while(i<size){
		atomicAdd( &temp[buffer[i]], 1);
		i+=offset;
	}
	__syncthreads();
if(threadIdx.x <256)
	atomicAdd( &(histo[threadIdx.x+memoffset]), temp[threadIdx.x] );
}

int main(void){

unsigned int host_histo[BLOCKS][256];
unsigned char *dev_buffer;
unsigned int *dev_histo;
hipEvent_t start, stop;
float elapsedTime;

//pedimos memoria
hipMalloc( (void**)&dev_buffer, SIZE );
hipMemcpy( dev_buffer, image, SIZE, hipMemcpyHostToDevice );
hipMalloc( (void**)&dev_histo,256 * BLOCKS * sizeof(long) );
hipMemset( dev_histo, 0, 256*BLOCKS *sizeof(long) );
//Medicion de tiempo de ejecucion
hipEventCreate(&start);
hipEventCreate(&stop);
// Start record
hipEventRecord(start, 0);
//KERNEL EXECUTION
histo_MultiBlock<<<BLOCKS,THREADS>>>(dev_buffer,SIZE,dev_histo);
// Stop event
hipEventRecord(stop, 0);
hipEventSynchronize(stop);
hipEventElapsedTime(&elapsedTime, start, stop); // that's our time!

//KERNEL HAS FINISHED 
printf("Kernel ejecutado en %3.2f ms\nCon %d bloques y %d hilos\n",elapsedTime,BLOCKS,THREADS);
//retorno de valores desde dev a host
hipMemcpy( host_histo, dev_histo, 256*BLOCKS*sizeof(int), hipMemcpyDeviceToHost );
/*
for(long i=0;i<256;i++){
	for(int j=1;j<BLOCKS;j++){
		printf("%d ",host_histo[j][i]);			
	}
}
printf("\n\n");*/
//unimos histogramas
for(long i=0;i<256;i++){
	for(int j=1;j<BLOCKS;j++){
		host_histo[0][i]+=host_histo[j][i];			
	}
}
//calculamos si el histograma es correcto en CPU, debemos obtener 0 (restamos en vez de sumar uno)
for(int i=0;i<256;i++){
	printf("%d ",host_histo[0][i]);
}
for(int i=0;i<SIZE;i++){
	host_histo[0][image[i]]--;
}
printf("\n\n");
//buscamos valores distintos de 0
for(int i=0;i<256;i++){
	if(host_histo[0][i]!=0){
		printf("valor %d incorrecto\n",i);
		break;
	}
}
//liberamos memoria
hipEventDestroy(start);
hipEventDestroy(stop);
hipFree(dev_histo);
hipFree(dev_buffer);

return 0;
}
