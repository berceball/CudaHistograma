#include "hip/hip_runtime.h"
#include <stdio.h>
#include "image.h"
#include <time.h>
#define SIZE 2048*1024

__global__ void histo_MonoBlock( unsigned char *buffer,long size,unsigned int *histo ) {

__shared__ unsigned int temp[256];
temp[threadIdx.x] = 0;
__syncthreads();

int i = threadIdx.x , offset = blockDim.x;
while (i < size) {
	atomicAdd( &temp[buffer[i]], 1);
	i += offset;
}
__syncthreads();

atomicAdd( &(histo[threadIdx.x]), temp[threadIdx.x] );

}


int main(void){

unsigned char *dev_buffer;
unsigned int *dev_histo;

//pedimos memoria
hipMalloc( (void**)&dev_buffer, SIZE );
hipMemcpy( dev_buffer, image, SIZE, hipMemcpyHostToDevice );

hipMalloc( (void**)&dev_histo,256 * sizeof(long) );
hipMemset( dev_histo, 0, 256*sizeof(int) );
//KERNEL EXECUTION
histo_MonoBlock<<<1,256>>>(dev_buffer,SIZE,dev_histo);
//KERNEL HAS FINISHED 
unsigned int host_histo[256];
//retorno de valores desde dev a host
hipMemcpy( host_histo, dev_histo, 256*sizeof(int),hipMemcpyDeviceToHost );
//calculamos si el histograma es correcto en CPU, debemos obtener 0 (restamos en vez de sumar uno)
for(int i=0;i<SIZE;i++){
	host_histo[image[i]]--;
}
//buscamos valores distintos de 0
for(int i=0;i<256;i++){
	if(host_histo[i]!=0) printf("valor %d incorrecto\n",i);
}
//liberamos memoria
hipFree(dev_histo);
hipFree(dev_buffer);

return 0;
}
